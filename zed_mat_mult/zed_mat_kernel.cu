#include "hip/hip_runtime.h"
#include "zed_mat_kernel.h"

// simple cuda matrix mul

#include "defines.h"
#include "stdio.h"

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) cutilBankChecker(((double*)&As[0][0]), (block_size * i + j))
#define BS(i, j) cutilBankChecker(((double*)&Bs[0][0]), (block_size * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif



void kerMatrixAdd_FromC(PREC_TYPE* d_c, const PREC_TYPE* d_a, const PREC_TYPE* d_b,int rows, int cols)
{
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	dim3 grid( 
		(rows / threads.x)+(rows%threads.x==0 ? 0 : 1),
		(cols / threads.y)+(cols%threads.y==0 ? 0 : 1));

	kerMatrixAdd<<< grid, threads >>>(d_c, d_a, d_b,rows,cols);	
}



void kerMatrixMul_FromC(PREC_TYPE* d_c, const PREC_TYPE* d_a, const PREC_TYPE* d_b,int a_rows, int a_cols,int b_rows, int b_cols)
{
	//dim3 grid((n+31)/32, (n+31)/32);

	// execute the kernel
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	//rows/BLOCK_THREADS_X+ (rows%BLOCK_THREADS_X == 0 ? 0:1

	dim3 grid( 
		(b_rows / threads.x)+(b_rows%threads.x==0 ? 0 : 1),
		(a_cols / threads.y)+(a_cols%threads.y==0 ? 0 : 1));

	//printf("block=%d x %d\n",threads.x,threads.y);
	//printf("grid=%d x %d\n",grid.x,grid.y);


	kerMatrixMul<<< grid, threads >>>(d_c, d_a, d_b,a_rows, a_cols,b_rows, b_cols);	

	//kerMatrixMul<<< grid, threads >>>(d_c, d_a, d_b, a_rows, b_rows);	
}

void kerMatrixMul_With_Scalar_FromC(PREC_TYPE* res, PREC_TYPE* src, PREC_TYPE s, int rows, int cols)
{
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid( 
		(rows / threads.x)+(rows%threads.x==0 ? 0 : 1),
		(cols / threads.y)+(cols%threads.y==0 ? 0 : 1));

	kerMatrixMulWithScalar<<< grid, threads >>>(res, src, s, rows, cols);	
}

void kerMatrixTranspose_FromC(PREC_TYPE* dst, PREC_TYPE* src, int src_rows, int src_cols)
{
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid( 
		(src_rows / threads.x)+(src_rows%threads.x==0 ? 0 : 1),
		(src_cols / threads.y)+(src_cols%threads.y==0 ? 0 : 1));

	kerMatrixTranspose<<< grid, threads >>>(dst,src,src_rows,src_cols);	
}


__global__ void kerMatrixAdd( PREC_TYPE* C, const PREC_TYPE* A, const PREC_TYPE* B,int rows, int cols)
{ 
	//current row
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	//current column
	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	//debug
	//printf("x=%d,y=%d\n",x,y);
	//C[y*b_rows + x] = 10.0;
	//return;

	if (x > rows-1)
	{
		return;
	}

	if (y > cols-1)
	{
		return;
	}

	C[y*cols + x] = A[y*cols + x] + B[y*cols + x];
}


//simplest - not optimizedw
//TODO: add optimization by using sub-matrices optimization + shared memory.
__global__ void kerMatrixMul( PREC_TYPE* C, const PREC_TYPE* A, const PREC_TYPE* B,int a_rows, int a_cols,int b_rows, int b_cols)
{ 
	//current row
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	//current column
	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	//debug
	//printf("x=%d,y=%d\n",x,y);
	//C[y*b_rows + x] = 10.0;
	//return;

	int c_rows = a_rows;
	int c_cols = b_cols;

	if (y > c_cols-1)
	{
		return;
	}

	if (x > c_rows-1)
	{
		return;
	}

	PREC_TYPE sum_val = 0;
	// each thread computes one element of the output matrix Pd.      
	for (int k = 0; k < a_cols; ++k) 
	{
		int a_ind = x*a_cols + k;
		int b_ind = k*b_cols + y;
		sum_val += A[a_ind] * B[b_ind];
	}

	// write back to the global memory
	C[x*c_cols + y] = sum_val;	
}

   /*/// 1. 2D Thread ID
   int tx = blockIdx.x * TILE_SIZE + threadIdx.x;
   int ty = blockIdx.y * TILE_SIZE + threadIdx.y;
 
   // value stores the element that is 
   // computed by the thread
   PREC_TYPE value = 0;
   for (int i = 0; i < wA; ++i)
   {
      PREC_TYPE elementA = A[ty * wA + i];
      PREC_TYPE elementB = B[i * wB + tx];
      value += elementA * elementB;
   }
 
   // Write the matrix to device memory each 
   // thread writes one element
   C[ty * wA + tx] = value;*/

/*__global__ void kerMatrixMul( PREC_TYPE* C, const PREC_TYPE* A, const PREC_TYPE* B, int wA, int wB)
{


	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;


	//check out of bounds
	if (bx == gridDim.x-1)
	{
		if (tx > 0)
		{
			return;
		}
	}

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd   = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep  = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep  = BLOCK_SIZE * wB;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	PREC_TYPE Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin;
			a <= aEnd;
			a += aStep, b += bStep)
	{

		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ PREC_TYPE As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ PREC_TYPE Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		AS(ty, tx) = A[a + wA * ty + tx];
		BS(ty, tx) = B[b + wB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
	#pragma unroll

		for (int k = 0; k < BLOCK_SIZE; ++k)
			Csub += AS(ty, k) * BS(k, tx);

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	int ind = c + wB * ty + tx;

	C[ind] = Csub;
}
*/


__global__ void kerMatrixMulWithScalar( PREC_TYPE* res, PREC_TYPE* src, PREC_TYPE s, int rows, int cols)
{
	//current row
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	//current column
	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	if (x > rows-1)
	{
		return;
	}

	if (y > cols-1)
	{
		return;
	}

	res[y*rows + x] = src[y*rows + x]*s;
}

__global__ void kerMatrixTranspose(PREC_TYPE* dst, PREC_TYPE* src, int src_rows, int src_cols)
{
	//current row
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	//current column
	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	if (x > src_rows-1)
	{
		return;
	}

	if (y > src_cols-1)
	{
		return;
	}

	dst[y*src_rows + x] = src[x*src_cols + y];
}