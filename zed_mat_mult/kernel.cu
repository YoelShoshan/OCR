#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "defines.h"
//#include <hip/hip_vector_types.h>
#include "zed_mat_mult.h"
#include "zed_mat_kernel.h"
#include <Windows.h>


//new
//#include <gpumatrix/CORE>
//using namespace gpumatrix;

//hipError_t addWithCuda(PREC_TYPE *c, const PREC_TYPE *a, const PREC_TYPE *b, size_t size);

/*#define WA 16   // Matrix A width
#define HA 16   // Matrix A height
#define WB 16   // Matrix B width
#define HB WA     // Matrix B height
#define WC WB     // Matrix C width
#define HC HA     // Matrix C height*/

/*

__global__ void kerAdd(PREC_TYPE *c, const PREC_TYPE *a, const PREC_TYPE *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(PREC_TYPE *c, const PREC_TYPE *a, const PREC_TYPE *b, size_t size)
{
    PREC_TYPE *dev_a = 0;
    PREC_TYPE *dev_b = 0;
    PREC_TYPE *dev_c = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(PREC_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(PREC_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(PREC_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(PREC_TYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(PREC_TYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    kerAdd<<<1, size>>>(dev_c, dev_a, dev_b);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(PREC_TYPE), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

void randomInit(PREC_TYPE* data, int size)
{
   for (int i = 0; i < size; ++i)
   {
		data[i] = rand() / (PREC_TYPE)RAND_MAX;
   }
}

void unitInit(PREC_TYPE* data, int width, int height)
{
   for (int i=0;i<width;++i)
   {
	   for (int j=0;j<height;j++)
	   {
		   if (i==j)
		   {
			   data[(width*j)+i] = 1.0;
		   } else
		   {
			   data[(width*j)+i] = 0.0;
		   }
	   }
   }
}

void zeroInit(PREC_TYPE* data, int size)
{
   for (int i = 0; i < size; ++i)
   {
		data[i] = 0.0;
   }
}*/

/*
int main()
{
	hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return -1;
    }

	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test_matMult_cuda failed!");
        return 1;
    }

	DWORD start = timeGetTime();

	ZedMatMult<PREC_TYPE,16,16,16,16> zed_mat_mult;

	randomInit(zed_mat_mult.h_a, zed_mat_mult.a_width*zed_mat_mult.a_height);
	unitInit(zed_mat_mult.h_b, zed_mat_mult.b_width,zed_mat_mult.b_height);
	zeroInit(zed_mat_mult.h_c, zed_mat_mult.c_width*zed_mat_mult.c_height);
	
	zed_mat_mult.mult();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
*/



