#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "bmp.h"
#include <assert.h>
using namespace std;

#define BLOCK_THREADS_X 16
#define BLOCK_THREADS_Y 16
#define THREADS_PER_BLOCK BLOCK_THREADS_X*BLOCK_THREADS_Y

__global__ void kernel_PassThrough(unsigned char *src, unsigned char *res, int width, int height)
{
	int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	int y = (blockIdx.y*blockDim.y) + threadIdx.y;

	int index = (y*width)+x;

    res[index] = src[index] + 30;
}

__global__ void kernel_EdgeDetection(unsigned char *src, unsigned char *res, int width, int height)
{
	int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	int y = (blockIdx.y*blockDim.y) + threadIdx.y;

	int index = (y*width)+x;
	int left_index = max(0, (y*width)+x-1);
	//int right = min(width-1, (y*width)+x+1);

	int diff = abs(int(src[index]) - int(src[left_index]));

	res[index] = unsigned char (diff);
    //res[index] = src[index] + 30;
}

#define KERNEL_DIM 7

__global__ void kernel_MedianThreshold(unsigned char *src, unsigned char *res, int width, int height)
{
	int x = (blockIdx.x*blockDim.x) + threadIdx.x;
	int y = (blockIdx.y*blockDim.y) + threadIdx.y;

	if (x > width-1 || y > height-1)
	{
		return;
	}

	int indices[KERNEL_DIM][KERNEL_DIM];

	int half_kernel_dim = KERNEL_DIM/2;
	int max_index = width*height-1;

	for(int i=0;i<KERNEL_DIM;i++)
	{
		for(int j=0;j<KERNEL_DIM;j++)
		{
			int _i = i-half_kernel_dim;
			int _j = j-half_kernel_dim;
			indices[i][j] = ( (y+_j)*width ) + (x+_i);
			indices[i][j] = max(0, indices[i][j]);
			indices[i][j] = min(max_index, indices[i][j]);
		}
	}

	int accum = 0;

	/*for(int i=0;i<KERNEL_DIM;i++)
	{
		for(int j=0;j<KERNEL_DIM;j++)
		{
			accum+= src[indices[i][j]];
		}
	}*/

	//perform binary search for the median

	int best_median = -1;
	int best_error = 0xFFFF;

	for (int c=0;c<256;c++)
	{
		int error = 0;

		for(int i=0;i<KERNEL_DIM;i++)
		{
			for(int j=0;j<KERNEL_DIM;j++)
			{
				unsigned char curr =  src[indices[i][j]];
				if (curr > c)
					error++;
				else
					error--;
			}
		}

		error = abs(error);
		if (error < best_error)
		{
			best_median = c;
			best_error = error;
		}
	}


	float fRes = float(accum) / float(KERNEL_DIM*KERNEL_DIM);
	unsigned char res_byte = (unsigned char) fRes;

	//res[indices[half_kernel_dim][half_kernel_dim]] = res_byte;

	if ( src[indices[half_kernel_dim][half_kernel_dim]] > best_median - 5)
	{
		res[indices[half_kernel_dim][half_kernel_dim]] = 255;
	} else
	{
		res[indices[half_kernel_dim][half_kernel_dim]] = 0;
	}
	
	/*int index = (y*width)+x;
	int left_index = max(0, (y*width)+x-1);
	//int right = min(width-1, (y*width)+x+1);

	int diff = abs(int(src[index]) - int(src[left_index]));

	res[index] = unsigned char (diff);*/
    //res[index] = src[index] + 30;
}



void checkForErrors()
{
	hipError_t err = hipGetLastError();
	if (hipSuccess!= err)
	{
		const char * err_str = hipGetErrorString(err);
		MessageBoxA(0,err_str,"Cuda Error!",0);
	}
}

void cuda_device_init(void)
{
	int ndev;
	hipGetDeviceCount(&ndev);
	hipDeviceSynchronize();
	printf("---- Cuda Devices Configuration ----\n");
	printf("There are %d GPUs.\n",ndev);
     
	for(int i=0;i<ndev;i++) {
	hipDeviceProp_t pdev;
	hipGetDeviceProperties(&pdev,i);
	hipDeviceSynchronize();
	printf("Name  : %s\n",pdev.name);
	printf("Capability  : %d %d\n",pdev.major,pdev.minor);
	printf("Memory Global: %d Mb\n",(pdev.totalGlobalMem+1024*1024)/1024/1024);
	printf("Memory Const : %d Kb\n",pdev.totalConstMem/1024);
	printf("Memory Shared: %d Kb\n",pdev.sharedMemPerBlock/1024);
	printf("Clock  : %.3f GHz\n",pdev.clockRate/1000000.0);
	printf("Processors  : %d\n",pdev.multiProcessorCount);
	printf("Cores  : %d\n",8*pdev.multiProcessorCount);
	printf("Warp  : %d\n",pdev.warpSize);
	printf("Max Thr/Blk  : %d\n",pdev.maxThreadsPerBlock);
	printf("Max Blk dimention Size : %d %d %d\n",pdev.maxThreadsDim[0],pdev.maxThreadsDim[1],pdev.maxThreadsDim[2]);
	printf("Max Grid dimention Size: %d %d %d\n",pdev.maxGridSize[0],pdev.maxGridSize[1],pdev.maxGridSize[2]);
}
}

int main(int args, char* argv[])
{
	if (args < 3)
	{
		printf("Usage: call with ""input.bmp"" ""output.bmp""\n");
		return 0;
	}



	cuda_device_init();

	unsigned int width=0;
	unsigned int height=0;
	unsigned int comps=0;
	
	//src image on host
	unsigned char* h_src = LoadBMP(argv[1],width,height,comps);
	assert(1==comps);

	//verify
	//SaveBMP_GreyScale(h_src,width,height,"C:/temp/cuda_thresholding/verify_bmp_lib.bmp");

	unsigned int pixels_num = width*height;
	int byte_size = sizeof(char)*pixels_num;
	//src image on device
	unsigned char* d_src = NULL;
	hipMalloc((void **)&d_src, byte_size);

	//result image on host
	unsigned char* h_res = new unsigned char[width*height];
	unsigned char* d_res = NULL;
	hipMalloc((void **)&d_res, byte_size);

	//copy src from host to device
	hipMemcpy(d_src, h_src, byte_size, hipMemcpyHostToDevice);

	checkForErrors();

	unsigned int blocks_num = pixels_num/THREADS_PER_BLOCK;

	dim3 block_threads(BLOCK_THREADS_X, BLOCK_THREADS_Y);
	//dim3 block_threads(10,10);

	//int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);

	//if it's not a perfect division, we need to add an extra block.
	//part of this thread block will work on pixels outside the image scope, we will make sure in the kernel that we don't write out-of-bounds
	dim3 grid_blocks(width/BLOCK_THREADS_X+ (width%BLOCK_THREADS_X == 0 ? 0:1), height/BLOCK_THREADS_Y + (height%BLOCK_THREADS_Y == 0 ? 0:1));
	//dim3 grid_blocks(10,10);

	printf("------------------------------\n");
	printf("Running (%d,%d,%d) blocks, each contains (%d,%d,%d) Threads.\n",
		block_threads.x,block_threads.y,block_threads.z,
		grid_blocks.x,grid_blocks.y,grid_blocks.z);

	kernel_MedianThreshold<<<grid_blocks,block_threads>>>(d_src,d_res, width, height);
	checkForErrors();

	//copy result from device to host
	hipMemcpy(h_res, d_res, byte_size, hipMemcpyDeviceToHost);

	SaveBMP_GreyScale(h_res,width,height,argv[2]);

    return 0;
}
